#include "hip/hip_runtime.h"
#include  "../mshadow/tensor.h"
#include  "../mshadow/tensor_container.h"

template <typename Dtype>
void im2col_cpu(const Dtype* data_im, const int channels,
    const int height, const int width, const int ksize, const int pad,
    const int stride, Dtype* data_col) {
  int height_col = (height + 2 * pad - ksize) / stride + 1;
  int width_col = (width + 2 * pad - ksize) / stride + 1;
  int channels_col = channels * ksize * ksize;
  for (int c = 0; c < channels_col; ++c) {
    int w_offset = c % ksize;
    int h_offset = (c / ksize) % ksize;
    int c_im = c / ksize / ksize;
    for (int h = 0; h < height_col; ++h) {
      for (int w = 0; w < width_col; ++w) {
        int h_pad = h * stride - pad + h_offset;
        int w_pad = w * stride - pad + w_offset;
        if (h_pad >= 0 && h_pad < height && w_pad >= 0 && w_pad < width)
          data_col[(c * height_col + h) * width_col + w] =
            data_im[(c_im * height + h_pad) * width + w_pad];
        else
          data_col[(c * height_col + h) * width_col + w] = 0;
      }
    }
  }
}


template <typename Dtype>
void col2im_cpu(const Dtype* data_col, const int channels,
    const int height, const int width, const int ksize, const int pad,
    const int stride, Dtype* data_im) {
  memset(data_im, 0, sizeof(Dtype) * height * width * channels);
  int height_col = (height + 2 * pad - ksize) / stride + 1;
  int width_col = (width + 2 * pad - ksize) / stride + 1;
  int channels_col = channels * ksize * ksize;
  for (int c = 0; c < channels_col; ++c) {
    int w_offset = c % ksize;
    int h_offset = (c / ksize) % ksize;
    int c_im = c / ksize / ksize;
    for (int h = 0; h < height_col; ++h) {
      for (int w = 0; w < width_col; ++w) {
        int h_pad = h * stride - pad + h_offset;
        int w_pad = w * stride - pad + w_offset;
        if (h_pad >= 0 && h_pad < height && w_pad >= 0 && w_pad < width)
          data_im[(c_im * height + h_pad) * width + w_pad] +=
              data_col[(c * height_col + h) * width_col + w];
      }
    }
  }
}
using namespace mshadow;
using namespace mshadow::expr;

template<typename xpu,int dim>
inline void Check( Tensor<xpu,dim>& xmat, Tensor<cpu,dim>& cmat ){
    TensorContainer<cpu,dim> txmat(false);
    txmat.Resize( xmat.shape );
    Copy(txmat, xmat);
    for( index_t  i =0; i < cmat.shape.Size(); ++ i ){
        if( fabs( txmat.dptr[i] - cmat.dptr[i])>1e-6 ){
            printf("erro, i=%d, a=%f, b=%fr\n", i, txmat.dptr[i], cmat.dptr[i]);
            exit(-1);
        }
    }
}

const int spad = 1;

template<typename xpu>
inline void test( int channels, int height, int width, int ksize, int stride ){
    int height_col = (height + 2* spad- ksize) / stride + 1;
    int width_col = (width +2*spad- ksize) / stride + 1;
    TensorContainer<cpu,3> cimg(false); cimg.Resize( Shape3( channels, height, width));
    TensorContainer<cpu,2> cmat(false); cmat.Resize( Shape2( channels * ksize*ksize, height_col*width_col ) );
    TensorContainer<xpu,3> ximg(false); ximg.Resize( cimg.shape );
    TensorContainer<xpu,2> xmat(false); xmat.Resize( cmat.shape );
    for( index_t  i =0; i < cimg.shape.Size(); ++ i ){
        cimg.dptr[i] = i;
    } 
    Copy( ximg, cimg );
    im2col_cpu( cimg.dptr, channels, height, width, ksize, spad, stride, cmat.dptr );
    //ximg = chpool<red::sum>( ximg, 2);

    xmat = unpack_patch2col( pad(ximg,spad) , ksize, stride );
    //xmat = unpack_patch2col( ximg, ksize, stride ) * 1.0f ;
    Check( xmat, cmat );
    col2im_cpu( cmat.dptr, channels, height, width, ksize, spad, stride, cimg.dptr ) ;
    Shape<3> pshape= ximg.shape; pshape[1]+=2*spad; pshape[0]+=2*spad;
    ximg = crop( pack_col2patch( xmat, pshape, ksize, stride ), ximg[0].shape );
    //ximg = F<op::identity>( pack_col2patch( xmat, ximg.shape, ksize, stride ));
    Check( ximg, cimg );
}

int main( int argc, char *argv[] ){
    InitTensorEngine( atoi(argv[1]) );
    for( int c = 1; c < 3; ++ c )
        for( int h = 5; h < 30; ++ h )
            for( int w = 6; w< 31; ++ w ){
                int kmax = 10;
                if( kmax > h ) kmax = h;
                if( kmax > w ) kmax = w;
                for( int ksize = 5; ksize < kmax; ++ ksize )
                    for( int stride = 1; stride < 8; ++ stride ){
                        test<cpu>( c,h,w,ksize, stride);
                        test<gpu>( c,h,w,ksize, stride);
                    }
            }
    ShutdownTensorEngine();
    return 0;
}
