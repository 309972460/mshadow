#include "hip/hip_runtime.h"
#include  "../mshadow/tensor.h"
#include  "../mshadow/tensor_container.h"

template <typename Dtype>
void im2col_cpu(const Dtype* data_im, const int channels,
    const int height, const int width, const int ksize, const int pad,
    const int stride, Dtype* data_col) {
  int height_col = (height + 2 * pad - ksize) / stride + 1;
  int width_col = (width + 2 * pad - ksize) / stride + 1;
  int channels_col = channels * ksize * ksize;
  for (int c = 0; c < channels_col; ++c) {
    int w_offset = c % ksize;
    int h_offset = (c / ksize) % ksize;
    int c_im = c / ksize / ksize;
    for (int h = 0; h < height_col; ++h) {
      for (int w = 0; w < width_col; ++w) {
        int h_pad = h * stride - pad + h_offset;
        int w_pad = w * stride - pad + w_offset;
        if (h_pad >= 0 && h_pad < height && w_pad >= 0 && w_pad < width)
          data_col[(c * height_col + h) * width_col + w] =
            data_im[(c_im * height + h_pad) * width + w_pad];
        else
          data_col[(c * height_col + h) * width_col + w] = 0;
      }
    }
  }
}


template <typename Dtype>
void col2im_cpu(const Dtype* data_col, const int channels,
    const int height, const int width, const int ksize, const int pad,
    const int stride, Dtype* data_im) {
  memset(data_im, 0, sizeof(Dtype) * height * width * channels);
  int height_col = (height + 2 * pad - ksize) / stride + 1;
  int width_col = (width + 2 * pad - ksize) / stride + 1;
  int channels_col = channels * ksize * ksize;
  for (int c = 0; c < channels_col; ++c) {
    int w_offset = c % ksize;
    int h_offset = (c / ksize) % ksize;
    int c_im = c / ksize / ksize;
    for (int h = 0; h < height_col; ++h) {
      for (int w = 0; w < width_col; ++w) {
        int h_pad = h * stride - pad + h_offset;
        int w_pad = w * stride - pad + w_offset;
        if (h_pad >= 0 && h_pad < height && w_pad >= 0 && w_pad < width)
          data_im[(c_im * height + h_pad) * width + w_pad] +=
              data_col[(c * height_col + h) * width_col + w];
      }
    }
  }
}
using namespace mshadow;
using namespace mshadow::expr;

template<typename xpu,int dim>
inline void Check( Tensor<xpu,dim>& xmat, Tensor<cpu,dim>& cmat ){
    TensorContainer<cpu,dim> txmat(false);
    txmat.Resize( xmat.shape );
    Copy(txmat, xmat);
    for( index_t  i =0; i < cmat.shape.Size(); ++ i ){
        if( fabs( txmat.dptr[i] - cmat.dptr[i])>1e-6 ){
            printf("erro, i=%d, a=%f, b=%fr\n", i, txmat.dptr[i], cmat.dptr[i]);
            exit(-1);
        }
    }
}

const int spad = 1;

template<typename xpu>
inline void testX( int num, int channels, int height, int width, int ksize, int stride ){
    int height_col = (height + 2* spad- ksize) / stride + 1;
    int width_col = (width +2*spad- ksize) / stride + 1;
    TensorContainer<cpu,4> cimg(false); cimg.Resize( Shape4( num, channels, height, width));
    TensorContainer<cpu,3> cmat(false); cmat.Resize( Shape3( num, channels*ksize*ksize, height_col*width_col ) );
    TensorContainer<xpu,4> ximg(false); ximg.Resize( cimg.shape );
    TensorContainer<xpu,2> xmat(false); xmat.Resize( Shape2( channels*ksize*ksize, height_col*width_col*num ) );
    TensorContainer<xpu,3> xtmat(false); xtmat.Resize( Shape3( channels*ksize*ksize, num, height_col*width_col ) );
    TensorContainer<xpu,3> xxmat(false); xxmat.Resize( cmat.shape );
    for( index_t  i =0; i < cimg.shape.Size(); ++ i ){
        cimg.dptr[i] = i;
    } 
    Copy( ximg, cimg );
    for( int n = 0; n < num; ++ n ){
        im2col_cpu( cimg[n].dptr , channels, height, width, ksize, spad, stride, cmat[n].dptr );
    }
    xmat = unpack_patch2col( pad(ximg,spad) , ksize, stride );
    xxmat = swapaxis<0,1>( swapaxis<0,2>( swapaxis<0,1>( reshape( xmat, xtmat.shape ) ) ));
    //Check( xxmat, cmat );
    for( int n = 0; n < num; ++ n ){
        col2im_cpu( cmat[n].dptr, channels, height, width, ksize, spad, stride, cimg[n].dptr ) ;
    }
    Shape<4> pshape= ximg.shape; pshape[1]+=2*spad; pshape[0]+=2*spad;
    ximg = crop( pack_col2patch( xmat, pshape, ksize, stride ), ximg[0][0].shape );
    Check( ximg, cimg );
}
#include <ctime>
int main( int argc, char *argv[] ){
    if( argc < 2 ){
        printf("Usage:<deviceid>\n"); exit(-1);
    }
    
    InitTensorEngine( atoi(argv[1]) );
    time_t start = time(NULL);
    for( int n = 2; n < 3; ++n )  
    for( int c = 1; c < 3; ++ c )
        for( int h = 5; h < 30; ++ h )
            for( int w = 25; w< 31; ++ w ){
                int kmax = 10;
                if( kmax > h ) kmax = h;
                if( kmax > w ) kmax = w;
                for( int ksize = 5; ksize < kmax; ++ ksize )
                    for( int stride = 1; stride < 8; ++ stride ){
                        testX<cpu>( n, c,h,w,ksize, stride);
                        testX<gpu>( n, c,h,w,ksize, stride);
                    }
            }    
    printf("all test passed, %lu sec\n", time(NULL) - start);
    ShutdownTensorEngine();
    return 0;
}
